
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void AddIntsCUDA(int* a, int* b){
	a[0] += b[0];
}

int main(){
	int a = 5;
	int b = 9;

	int *d_a;
	int *d_b;

	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCUDA <<<1, 1 >>>(d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

	cout << "the answer is: " << a << endl;

	return 0;
}

//Need Error Checking i.e. DeAllocation in case of failure. 